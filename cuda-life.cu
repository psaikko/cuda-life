
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <chrono>
#include <thread>
#include <iostream>

using namespace std;

const int W = 1000;
const int H = 1000;

__global__
void compute_transition(const bool *const current, bool *const next) { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int y = index; y < H; y += stride) {
        for (int x = 0; x < W; ++x) {
            int i = y * W + x;
            int n = 0;
            for (int yd = -1; yd <= 1; yd++) {
                int y_ = (y + H + yd) % H;
                for (int xd = -1; xd <= 1; xd++) {
                    if (yd == 0 && xd == 0) continue;
                    int x_ = (x + W + xd) % W;
                    n += current[y_ * W + x_];
                }
            }

            if (current[i]) {
                next[i] = (n == 2 || n == 3);
            } else {
                next[i] = (n == 3);
            }
        }
    }
}

void print_grid(const bool *const grid) {
    for (int y = 0; y < H; ++y) {
        for (int x = 0; x < W; ++x) {
            cout << (grid[y*W+x] ? "#" : " ");
        }
        cout << endl;
    }
}

int main() {
    bool *current, *next;

    hipMallocManaged(&current, sizeof(bool) * W * H);
    hipMallocManaged(&next, sizeof(bool) * W * H);

    for (int i = 0; i < W*H; ++i)
        current[i] = rand() % 2;

    auto start_time = chrono::high_resolution_clock::now();

    int blockSize = 256;
    int nBlocks = (W*H + blockSize - 1) / blockSize;

    for (int i = 0; i < 100; ++i) {
        compute_transition<<<nBlocks, blockSize>>>(current, next);
        swap(current, next);
    }

    hipDeviceSynchronize();

    auto end_time = chrono::high_resolution_clock::now();
    auto ms = chrono::duration_cast<chrono::milliseconds>(end_time - start_time);

    cout << ms.count() << "ms" << endl;

    hipFree(next);
    hipFree(current);
}